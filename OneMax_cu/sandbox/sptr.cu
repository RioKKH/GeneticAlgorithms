#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <helper_functions.h>

__global__ void gpuKernel1(int N, float* C, float* A)
{
	for (int i = 0; i < N; i++) {
		C[i] = A[i];
		printf("%d\n", i);
	}
}

__global__ void gpuKernel2(int N, float* C)
{
	for (int i = 0; i < N; i++) {
		printf("gpuKernel2 %d, %d, %d, %f\n",
				threadIdx.x, blockIdx.x, blockDim.x, C[i]);
	}
}

class individual
{
public:
	std::shared_ptr<int> chromosome;
}


